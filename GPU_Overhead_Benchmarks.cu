#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include "CUDA_Wrapper.hpp"

__global__ void null_kernel(void) {
}

__global__ void copy_buffer(float* srcBuffer, float* dstBuffer)
{
	uint32_t idx = threadIdx.x;

	dstBuffer[idx] = srcBuffer[idx];
}

namespace CUDA_Kernels
{

	void nullKernelExecute()
	{
		null_kernel << <1, 1>> > ();
	}
	void copyBufferExecute(size_t aN, float* srcBuffer, float* dstBuffer)
	{
		float* d_srcBuffer;
		float* d_dstBuffer;
		hipMalloc((void**)&d_srcBuffer, sizeof(float) * aN);
		hipMalloc((void**)&d_dstBuffer, sizeof(float) * aN);

		hipMemcpy(d_srcBuffer, srcBuffer, sizeof(float) * aN, hipMemcpyHostToDevice);
		hipMemcpy(d_dstBuffer, dstBuffer, sizeof(float) * aN, hipMemcpyHostToDevice);

		dim3 threadsPerBlock(aN);
		copy_buffer << <1, aN >> > (d_srcBuffer, d_dstBuffer);

		hipMemcpy(dstBuffer, d_dstBuffer, sizeof(float) * aN, hipMemcpyDeviceToHost);

		hipFree(d_srcBuffer);
		hipFree(d_dstBuffer);
	}

}

//void wrapper(void)
//{
//	null_kernel<<<1,1>>>();
//	uint32_t N = 1;
//	float* d_out;
//	hipMalloc((void**)&d_out, sizeof(float) * N);
//
//	
//
//	float* checkWorking = (float*)malloc(sizeof(float) * N);
//	hipMemcpy(d_out, checkWorking, sizeof(float) * N, hipMemcpyHostToDevice);
//	//*checkWorking = 0.0;
//	test_kernel << <1, 1 >> > (d_out);
//
//	// Transfer data back to host memory
//	hipMemcpy(checkWorking, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
//	printf("Hello, world! %f", checkWorking[0]);
//}